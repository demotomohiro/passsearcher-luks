/*
Copyright 2016 Tomohiro Matsumoto

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include "hmacSha1Cuda.hpp"
#include "sha1Block.hpp"
#include "mdCuda.hpp"
#include <cassert>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

struct myCudaDeletor
{
	void operator()(void* ptr)
	{
		hipFree(ptr);
	}
};

template<typename T>
using myCudaUniquePtr = std::unique_ptr<T, myCudaDeletor>;

template<typename T>
__host__ myCudaUniquePtr<T[]> makeCudaUniquePtr(const size_t size)
{
	T* ptr;
	hipMalloc(&ptr, sizeof(T) * size);
	return myCudaUniquePtr<T[]>(ptr, myCudaDeletor());
}

class AF
{
public:

	AF(size_t blockBytes, size_t numBlocks, size_t numKeys):
		blockSize(blockBytes/sizeof(ui32)), numBlocks(numBlocks), numKeys(numKeys),
		AfKeys(new ui32[blockBytes/sizeof(ui32) * numBlocks * numKeys]),
		deviceAfKeys(makeCudaUniquePtr<ui32>(blockBytes/sizeof(ui32) * numBlocks * numKeys))
	{
		assert(blockBytes%sizeof(ui32) == 0);
	}

	void setAfKey(size_t idx, const char* src)
	{
		char* dst = reinterpret_cast<char*>(&AfKeys[blockSize * numBlocks * idx]);
		memcpy(dst, src, sizeof(ui32) * blockSize * numBlocks);
	}

	void AFMerge(thrust::host_vector<sha1InputBlock>&	output)
	{
		hipMemcpy(deviceAfKeys.get(), AfKeys.get(), sizeof(ui32) * blockSize * numBlocks * numKeys, hipMemcpyHostToDevice);
		cuda::AFMerge(deviceAfKeys.get(), blockSize, numBlocks, numKeys, output);
	}

	AF& operator=(const AF&);

	const size_t blockSize;
	const size_t numBlocks;
	const size_t numKeys;

	std::unique_ptr<ui32[]>	AfKeys;
	myCudaUniquePtr<ui32[]>	deviceAfKeys;
};

struct hmacSha1CudaPrivate
{
	hmacSha1CudaPrivate()
	{
	}

	hmacSha1CudaPrivate(size_t blockBytes, size_t numBlocks, size_t numKeys):
		pAF(new AF(blockBytes, numBlocks, numKeys))
	{
	}

	thrust::host_vector<sha1InputBlock>		hostKey;
	sha1InputBlock							hostIn;
	thrust::host_vector<sha1Output>			hostOut;
	thrust::host_vector<derivedKey>			hostPbkdf2Out;

	std::unique_ptr<AF>						pAF;
};

hmacSha1Cuda::hmacSha1Cuda():priv(new hmacSha1CudaPrivate)
{
}

hmacSha1Cuda::hmacSha1Cuda(size_t blockBytes, size_t numBlocks, size_t numKeys):
	priv(new hmacSha1CudaPrivate(blockBytes, numBlocks, numKeys))
{
}

hmacSha1Cuda::~hmacSha1Cuda()
{
}

size_t hmacSha1Cuda::getMaxKeyLength()
{
	return sha1InputBlock::blockBytes - 1;
}

size_t hmacSha1Cuda::getBatchSize()
{
	int val;
	hipError_t e = hipDeviceGetAttribute(&val, hipDeviceAttributeMultiprocessorCount, 0);
	if(e != hipSuccess)
	{
		return 0;
	}

	return val*256;
}

const unsigned char* hmacSha1Cuda::getHash
(
	const char* pass, size_t passLen,
	const char* salt, size_t saltLen
)
{
	thrust::host_vector<sha1InputBlock>		hostKey(1);
	sha1InputBlock							hostIn;
	hostKey[0].setKey(pass, passLen);
	hostIn.set(salt, saltLen);
	hostIn.setInputLengthHmac(saltLen);
	assert(hostIn.getLengthHmac() == saltLen);
//	hostIn[0].print();
	cuda::hmacSha1(hostKey, hostIn, priv->hostOut);

	return priv->hostOut[0].data();
}

const unsigned char* hmacSha1Cuda::getPbkdf2
(
	const char* pass, size_t passLen,
	const char* salt, size_t saltLen,
	unsigned int numIteration
)
{
	resize(1);
	clearInput();
	clearKey(0);
	memcpy(key(0), pass, passLen);
	memcpy(input(), salt, saltLen);
	setInputLength(saltLen);
	assert(checkInput());
	assert(getLength() == saltLen);
	transformPbkdf2(numIteration);
	return outputPbkdf2(0);
}

void hmacSha1Cuda::resize(size_t n)
{
	priv->hostKey.resize(n);
}

void hmacSha1Cuda::clearInput()
{
	priv->hostIn.clear();
}

void hmacSha1Cuda::clearKey(size_t idx)
{
	priv->hostKey[idx].clear();
}

char* hmacSha1Cuda::key(size_t idx)
{
	return priv->hostKey[idx].data();
}

char* hmacSha1Cuda::input()
{
	return priv->hostIn.data();
}

void hmacSha1Cuda::setInputLength(size_t length)
{
	priv->hostIn.setInputLengthHmac(length);
}

bool hmacSha1Cuda::checkInput()
{
	return priv->hostIn.checkInputHmac();
}

void hmacSha1Cuda::setAfKey(size_t idx, const char* src)
{
	assert(priv->pAF);
	priv->pAF->setAfKey(idx, src);
}

void hmacSha1Cuda::transform()
{
	cuda::hmacSha1(priv->hostKey, priv->hostIn, priv->hostOut);
}

void hmacSha1Cuda::AFMerge()
{
	assert(priv->pAF);

	priv->pAF->AFMerge(priv->hostKey);
}

void hmacSha1Cuda::transformPbkdf2(unsigned int numIteration)
{
	cuda::pbkdf2(priv->hostKey, priv->hostIn, numIteration, priv->hostPbkdf2Out);
}

size_t hmacSha1Cuda::getLength()
{
	return priv->hostIn.getLengthHmac();
}

const unsigned char* hmacSha1Cuda::output(size_t idx) const
{
	return priv->hostOut[idx].data();
}

const unsigned char* hmacSha1Cuda::outputPbkdf2(size_t idx) const
{
	return priv->hostPbkdf2Out[idx].data();
}

struct sha1CudaPrivate
{
	thrust::host_vector<sha1InputBlock>		hostIn;
	thrust::host_vector<sha1Output>			hostOut;
};

sha1Cuda::sha1Cuda():priv(new sha1CudaPrivate)
{
}

sha1Cuda::~sha1Cuda()
{
}

const unsigned char* sha1Cuda::getHash
(
	const char* pass, size_t passLen,
	const char* salt, size_t saltLen
)
{
	thrust::host_vector<sha1InputBlock>		hostIn(1);
	hostIn[0].set(salt, saltLen);
	assert(hostIn[0].getLength() == saltLen);
//	hostIn[0].print();
	cuda::sha1(hostIn, priv->hostOut);

	return priv->hostOut[0].data();
}

void sha1Cuda::resize(size_t n)
{
	priv->hostIn.resize(n);
}

void sha1Cuda::clear(size_t idx)
{
	priv->hostIn[idx].clear();
}

char* sha1Cuda::input(size_t idx)
{
	return priv->hostIn[idx].data();
}

void sha1Cuda::setInputLength(size_t idx, size_t length)
{
	priv->hostIn[idx].setInputLength(length);
}

void sha1Cuda::transform()
{
	cuda::sha1(priv->hostIn, priv->hostOut);
}

size_t sha1Cuda::getLength(size_t idx)
{
	return priv->hostIn[idx].getLength();
}

const unsigned char* sha1Cuda::output(size_t idx) const
{
	return priv->hostOut[idx].data();
}

